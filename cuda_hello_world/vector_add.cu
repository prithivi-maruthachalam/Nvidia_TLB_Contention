#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 100000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    out[i] = a[i] + b[i];
}

int main() {
    float *a, *b, *out;

    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = (float)(1.0f * i);
        b[i] = (float)(2.0f * i);
    }

    clock_t start, end;
    start = clock();
    vector_add<<<1, 1>>>(out, a, b, N);
    end = clock();

    double time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Function took %f seconds\n", time_used);
}